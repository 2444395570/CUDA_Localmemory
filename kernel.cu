#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

#define N 5

__global__ void gpu_local_memory(int d_in) {
	int t_local;
	t_local = d_in * threadIdx.x;
	printf("Value of local variable in current thread is:%d\n", t_local);
}

int main(void) {
	printf("Use of local Memory on GPU:\n");
	gpu_local_memory << <1, N >> > (5);
	hipDeviceSynchronize();
	return 0;
}

/*
�����ڴ�ͼĴ����Ѷ�ÿ���̶߳���Ψһ�ġ��Ĵ�����ÿ���߳̿��õ����洢�������ں���ʹ�õı����ڼĴ������з�
���µ�ʱ�򣬽���ʹ�ñ����ڴ�洢���ǣ���мĴ��������
ʹ�ñ����ڴ������������һ���ǼĴ���������һ����ĳЩ����������ܷ��ڼĴ����У������һ���ֲ�������±����
����������ʱ�򡣻����Ͽ��Խ������ڴ濴��ÿ���̵߳�Ψһ��ȫ���ڴ沿�֡���ȼĴ����ѣ������ڴ�Ҫ���ܶࡣ��Ȼ
�����ڴ�ͨ��L1�����L2��������˻��壬���ǼĴ���������ܻ�Ӱ����ĳ�������ܡ�

*/